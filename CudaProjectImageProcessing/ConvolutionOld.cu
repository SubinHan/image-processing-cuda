#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <cmath>
#include <algorithm>
#include <iostream>
#include "hipfft/hipfft.h"
#include "hipfft/hipfftXt.h"

#include "ConvolutionOld.cuh"

__global__ void write_data_to_image(uint8_t* image, int bpp, int offset, int size, hipfftReal* data);
__global__ void collect_data(uint8_t* image_in, int bpp, int offset, int size, hipfftReal* image_out);
__global__ void pad_kernel(
	hipfftReal* kernel_input,
	const int image_width,
	const int image_height,
	const int kernel_size,
	hipfftReal* kernel_output);
__global__ void pointwise_product(hipfftComplex* a, hipfftComplex* b, int size, float weight);

static const char* _cudaGetErrorEnum(hipfftResult error) {
	switch (error) {
	case HIPFFT_SUCCESS:
		return "HIPFFT_SUCCESS";

	case HIPFFT_INVALID_PLAN:
		return "HIPFFT_INVALID_PLAN";

	case HIPFFT_ALLOC_FAILED:
		return "HIPFFT_ALLOC_FAILED";

	case HIPFFT_INVALID_TYPE:
		return "HIPFFT_INVALID_TYPE";

	case HIPFFT_INVALID_VALUE:
		return "HIPFFT_INVALID_VALUE";

	case HIPFFT_INTERNAL_ERROR:
		return "HIPFFT_INTERNAL_ERROR";

	case HIPFFT_EXEC_FAILED:
		return "HIPFFT_EXEC_FAILED";

	case HIPFFT_SETUP_FAILED:
		return "HIPFFT_SETUP_FAILED";

	case HIPFFT_INVALID_SIZE:
		return "HIPFFT_INVALID_SIZE";

	case HIPFFT_UNALIGNED_DATA:
		return "HIPFFT_UNALIGNED_DATA";

	case HIPFFT_INCOMPLETE_PARAMETER_LIST:
		return "HIPFFT_INCOMPLETE_PARAMETER_LIST";

	case HIPFFT_INVALID_DEVICE:
		return "HIPFFT_INVALID_DEVICE";

	case HIPFFT_PARSE_ERROR:
		return "HIPFFT_PARSE_ERROR";

	case HIPFFT_NO_WORKSPACE:
		return "HIPFFT_NO_WORKSPACE";

	case HIPFFT_NOT_IMPLEMENTED:
		return "HIPFFT_NOT_IMPLEMENTED";

	case CUFFT_LICENSE_ERROR:
		return "CUFFT_LICENSE_ERROR";

	case HIPFFT_NOT_SUPPORTED:
		return "HIPFFT_NOT_SUPPORTED";
	}

	return "<unknown>";
}

static const char* _cudaGetErrorEnum(hipError_t error) {
	return hipGetErrorName(error);
}

template <typename T>
void check(T result, char const* const func, const char* const file,
	int const line) {
	if (result) {
		fprintf(stderr, "CUDA error at %s:%d code=%d(%s) \"%s\" \n", file, line,
			static_cast<unsigned int>(result), _cudaGetErrorEnum(result), func);
		exit(EXIT_FAILURE);
	}
}
#define checkCudaErrors(val) check((val), #val, __FILE__, __LINE__)

__global__ void collect_data(uint8_t* image_in, int bpp, int offset, int size, hipfftReal* image_out)
{
	unsigned int thread_index = blockIdx.x * blockDim.x + threadIdx.x;

	if (thread_index >= size)
		return;

	image_out[thread_index] = image_in[thread_index * bpp + offset];
}

__global__ void write_data_to_image(uint8_t* image, int bpp, int offset, int size, hipfftReal* data)
{
	unsigned int thread_index = blockIdx.x * blockDim.x + threadIdx.x;

	if (thread_index > size)
		return;

	if (data[thread_index] >= 255.f)
		data[thread_index] = 255.f;
	image[thread_index * bpp + offset] = data[thread_index];
}

__global__ void pad_kernel(hipfftReal* kernel_input, const int image_width, const int image_height, const int kernel_size, hipfftReal* kernel_output)
{
	unsigned int thread_index = blockIdx.x * blockDim.x + threadIdx.x;
	const int size = image_width * image_height;
	
	if (thread_index >= size)
		return;

	const int min_radius = kernel_size / 2;
	const int max_radius = kernel_size - min_radius;

	const int x = thread_index % image_width;
	const int y = thread_index / image_width;

	const bool is_x_left = x < max_radius;
	const bool is_x_right = x >= image_width - min_radius;
	const bool is_y_up = y < max_radius;
	const bool is_y_down = y >= image_height - min_radius;

	//printf("[%d] (%d, %d) (%d %d %d %d)\n", thread_index, x, y, is_x_left, is_x_right, is_y_up, is_y_down);
	//printf("[%d] [%d %d %d %d]\n", thread_index, is_x_left && is_y_up, is_x_right && is_y_up, is_x_left && is_y_down, is_x_right && is_y_down);
	//printf("[%d] [%d]\n", thread_index, (min_radius + y) * kernel_size + (x - (image_width - min_radius)));
	if (is_x_left && is_y_up)
	{
		int offset = (min_radius + y) * kernel_size + (min_radius + x);
		//printf("[%d] %d\n", thread_index, offset);
		kernel_output[thread_index] = kernel_input[offset];
		return;
	}

	if (is_x_right && is_y_up)
	{
		int offset = (min_radius + y) * kernel_size + (x - (image_width - min_radius));
		//printf("[%d] %d\n", thread_index, offset);
		kernel_output[thread_index] = kernel_input[offset];
		return;
	}

	if (is_x_left && is_y_down)
	{
		int offset = (y - (image_height - min_radius)) * kernel_size + (min_radius + x);
		//printf("[%d] %d\n", thread_index, offset);
		kernel_output[thread_index] = kernel_input[offset];
		return;
	}

	if (is_x_right && is_y_down)
	{
		int offset = (y - (image_height - min_radius)) * kernel_size + (x - (image_width - min_radius));
		//printf("[%d] %d\n", thread_index, offset);
		kernel_output[thread_index] = kernel_input[offset];
		return;
	}

	//printf("[%d] 0\n", thread_index);
	kernel_output[thread_index] = 0.f;
}

__global__ void pointwise_product(hipfftComplex* a, hipfftComplex* b, int size, float weight)
{
	unsigned thread_index = blockIdx.x * blockDim.x + threadIdx.x;
	if (thread_index >= size)
		return;

	//printf("[%d] (%f, %f) * (%f, %f) = \n", thread_index, a[thread_index].x, a[thread_index].y, b[thread_index].x, b[thread_index].y);

	float a_real_original = a[thread_index].x;
	a[thread_index].x = a[thread_index].x * b[thread_index].x - a[thread_index].y * b[thread_index].y;
	a[thread_index].y = a_real_original * b[thread_index].y + a[thread_index].y * b[thread_index].x;
	//printf("[%d] (%f, %f) = \n", thread_index, a[thread_index].x, a[thread_index].y);
	a[thread_index].x *= weight;
	a[thread_index].y *= weight;

	//printf("[%d] = (%f, %f)\n", thread_index, a[thread_index].x, a[thread_index].y);
}

__global__ void print_2d_real(hipfftReal* d_arr, int width, int height)
{
	for (int i = 0; i < height; i++)
	{
		//printf("%d: ", i);
		for (int j = 0; j < width; j++)
		{
			//printf("%4d", j);
			printf("%3.1f ", d_arr[i * width + j]);
		}
		printf("\n");
	}
	printf("\n");
}

void ConvolutionCalculator::convolution(
	const uint8_t* image_in,
	const float* kernel,
	const int image_width,
	const int image_height,
	const int kernel_width,
	const int kernel_height,
	const int bpp,
	uint8_t* image_out
)
{
	const int image_size = image_width * image_height * bpp;
	const int image_real_size = image_width * image_height;
	const int kernel_size = kernel_width * kernel_height;

	const int complex_size = image_width * (image_height / 2 + 1);

	uint8_t* d_int8_image = nullptr;
	checkCudaErrors(hipMalloc((void**)&d_int8_image, image_size * sizeof(uint8_t)));
	//create_in_device<uint8_t> << <1, 1 >> > (&d_int8_image, image_size);
	checkCudaErrors(hipMemcpy(d_int8_image, image_in, image_size * sizeof(uint8_t), hipMemcpyHostToDevice));


	hipfftReal* d_real_kernel = nullptr;
	checkCudaErrors(hipMalloc((void**)&d_real_kernel, kernel_size * sizeof(hipfftReal)));
	//create_in_device<hipfftReal> << <1, 1 >> > (&d_real_kernel, kernel_size);

	checkCudaErrors(hipMemcpy(d_real_kernel, kernel, kernel_size * sizeof(hipfftReal), hipMemcpyHostToDevice));

	hipfftReal* d_real_kernel_padded = nullptr;
	checkCudaErrors(hipMalloc((void**)&d_real_kernel_padded, image_real_size * sizeof(hipfftReal)));
	//create_in_device<hipfftReal> << <1, 1 >> > (&d_real_kernel_padded, image_real_size);

	checkCudaErrors(hipDeviceSynchronize());

	int blocksx = ceil((image_real_size) / 256.0f);
	dim3 threads(256);
	dim3 grid(blocksx);

	//printf("grid: %d, threads: %d\n", grid.x, threads.x);
	pad_kernel << <grid, threads >> > (d_real_kernel, image_width, image_height, kernel_width, d_real_kernel_padded);
	
	checkCudaErrors(hipDeviceSynchronize());

	//print_2d_real << <1, 1 >> > (d_real_kernel_padded, image_width, image_height);

	checkCudaErrors(hipDeviceSynchronize());
	hipfftHandle plan_kernel_to_complex;
	checkCudaErrors(hipfftPlan2d(&plan_kernel_to_complex, image_height, image_width, HIPFFT_R2C));

	hipfftComplex* d_complex_kernel = nullptr;
	checkCudaErrors(hipMalloc((void**)&d_complex_kernel, complex_size * sizeof(hipfftComplex)));
	//create_in_device<hipfftComplex> << <1, 1 >> > (&d_complex_kernel, complex_size);
	hipfftExecR2C(plan_kernel_to_complex, d_real_kernel_padded, d_complex_kernel);

	for (int i = 0; i < bpp; i++)
	{
		hipfftReal* d_real_image = nullptr;

		checkCudaErrors(hipMalloc((void**)&d_real_image, image_real_size * sizeof(hipfftReal)));
		//create_in_device<hipfftReal> << <1, 1 >> > (&d_real_image, image_real_size);

		collect_data<<<grid, threads>>>(d_int8_image, bpp, i, image_real_size, d_real_image);

		checkCudaErrors(hipDeviceSynchronize());
		//print_2d_real << <1, 1 >> > (d_real_image, image_width, image_height);

		hipfftComplex* d_complex_image = nullptr;

		checkCudaErrors(hipMalloc((void**)&d_complex_image, complex_size * sizeof(hipfftComplex)));
		checkCudaErrors(hipMemset(d_complex_image, 0, complex_size * sizeof(hipfftComplex)));
		checkCudaErrors(hipDeviceSynchronize());
		//create_in_device<hipfftComplex> << <1, 1 >> > (&d_complex_image, complex_size);

		hipfftHandle plan_image_to_complex, plan_result_to_real;
		checkCudaErrors(hipfftPlan2d(&plan_image_to_complex, image_height, image_width, HIPFFT_R2C));
		checkCudaErrors(hipfftPlan2d(&plan_result_to_real, image_height, image_width, HIPFFT_C2R));

		checkCudaErrors(hipfftExecR2C(plan_image_to_complex, d_real_image, d_complex_image));
		checkCudaErrors(hipDeviceSynchronize());

		int multiplication_blocksx = ceil(complex_size / 256.0f);
		dim3 multiplication_threads(256);
		dim3 multiplication_grid(multiplication_blocksx);
		//std::cout << multiplication_grid.x << multiplication_grid.y << multiplication_grid.z << multiplication_threads.x << multiplication_threads.y << multiplication_threads.z;
		printf("%f", 1.0f / (image_width * image_height));
		pointwise_product << <multiplication_grid, multiplication_threads >> > (d_complex_image, d_complex_kernel, complex_size, 1.0f / (image_width * image_height));
		checkCudaErrors(hipDeviceSynchronize());

		checkCudaErrors(hipfftExecC2R(plan_result_to_real, d_complex_image, d_real_image));
		checkCudaErrors(hipDeviceSynchronize());

		//print_2d_real << <1, 1 >> > (d_real_image, image_width, image_height);

		write_data_to_image << <grid, threads >> > (d_int8_image, bpp, i, image_real_size, d_real_image);
		checkCudaErrors(hipDeviceSynchronize());

		hipFree(d_real_image);
		hipFree(d_complex_image);
		/*destroy_in_device << <1, 1 >> > (d_real_image);
		destroy_in_device << <1, 1 >> > (d_complex_image);*/

		hipfftDestroy(plan_image_to_complex);
		hipfftDestroy(plan_result_to_real);
	}
	hipfftDestroy(plan_kernel_to_complex);

	checkCudaErrors(hipMemcpy(image_out, d_int8_image, image_size * sizeof(uint8_t), hipMemcpyDeviceToHost));

	hipFree(d_real_kernel);
	hipFree(d_real_kernel_padded);
	hipFree(d_complex_kernel);
	hipFree(d_int8_image);
	//destroy_in_device << <1, 1 >> > (d_real_kernel);
	//destroy_in_device << <1, 1 >> > (d_real_kernel_padded);
	//destroy_in_device << <1, 1 >> > (d_complex_kernel);
	//destroy_in_device << <1, 1 >> > (d_int8_image);

	checkCudaErrors(hipPeekAtLastError());
}
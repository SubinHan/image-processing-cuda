#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <cmath>
#include <algorithm>
#include <iostream>
#include "hipfft/hipfft.h"
#include "hipfft/hipfftXt.h"

#include "ConvolutionOld.cuh"

__global__ void write_data_to_image(uint8_t* image, int bpp, int offset, int size, hipfftDoubleReal* data);
__global__ void collect_data(uint8_t* image_in, int bpp, int offset, int size, hipfftDoubleReal* image_out);
__global__ void pad_kernel(
	hipfftDoubleReal* kernel_input,
	const int image_width,
	const int image_height,
	const int kernel_size,
	hipfftDoubleReal* kernel_output);
__global__ void pointwise_product(hipfftDoubleComplex* a, hipfftDoubleComplex* b, int size, double weight);

static const char* _cudaGetErrorEnum(hipfftResult error) {
	switch (error) {
	case HIPFFT_SUCCESS:
		return "HIPFFT_SUCCESS";

	case HIPFFT_INVALID_PLAN:
		return "HIPFFT_INVALID_PLAN";

	case HIPFFT_ALLOC_FAILED:
		return "HIPFFT_ALLOC_FAILED";

	case HIPFFT_INVALID_TYPE:
		return "HIPFFT_INVALID_TYPE";

	case HIPFFT_INVALID_VALUE:
		return "HIPFFT_INVALID_VALUE";

	case HIPFFT_INTERNAL_ERROR:
		return "HIPFFT_INTERNAL_ERROR";

	case HIPFFT_EXEC_FAILED:
		return "HIPFFT_EXEC_FAILED";

	case HIPFFT_SETUP_FAILED:
		return "HIPFFT_SETUP_FAILED";

	case HIPFFT_INVALID_SIZE:
		return "HIPFFT_INVALID_SIZE";

	case HIPFFT_UNALIGNED_DATA:
		return "HIPFFT_UNALIGNED_DATA";

	case HIPFFT_INCOMPLETE_PARAMETER_LIST:
		return "HIPFFT_INCOMPLETE_PARAMETER_LIST";

	case HIPFFT_INVALID_DEVICE:
		return "HIPFFT_INVALID_DEVICE";

	case HIPFFT_PARSE_ERROR:
		return "HIPFFT_PARSE_ERROR";

	case HIPFFT_NO_WORKSPACE:
		return "HIPFFT_NO_WORKSPACE";

	case HIPFFT_NOT_IMPLEMENTED:
		return "HIPFFT_NOT_IMPLEMENTED";

	case CUFFT_LICENSE_ERROR:
		return "CUFFT_LICENSE_ERROR";

	case HIPFFT_NOT_SUPPORTED:
		return "HIPFFT_NOT_SUPPORTED";
	}

	return "<unknown>";
}

static const char* _cudaGetErrorEnum(hipError_t error) {
	return hipGetErrorName(error);
}

template <typename T>
void check(T result, char const* const func, const char* const file,
	int const line) {
	if (result) {
		fprintf(stderr, "CUDA error at %s:%d code=%d(%s) \"%s\" \n", file, line,
			static_cast<unsigned int>(result), _cudaGetErrorEnum(result), func);
		exit(EXIT_FAILURE);
	}
}
#define checkCudaErrors(val) check((val), #val, __FILE__, __LINE__)

__global__ void collect_data(uint8_t* image_in, int bpp, int offset, int size, hipfftDoubleReal* image_out)
{
	unsigned int thread_index = blockIdx.x * blockDim.x + threadIdx.x;

	if (thread_index >= size)
		return;

	image_out[thread_index] = image_in[thread_index * bpp + offset];
}

__global__ void write_data_to_image(uint8_t* image, int bpp, int offset, int size, hipfftDoubleReal* data)
{
	unsigned int thread_index = blockIdx.x * blockDim.x + threadIdx.x;

	if (thread_index > size)
		return;

	if (data[thread_index] >= 255.f)
		data[thread_index] = 255.f;
	image[thread_index * bpp + offset] = data[thread_index];
}

__global__ void pad_kernel(hipfftDoubleReal* kernel_input, const int image_width, const int image_height, const int kernel_size, hipfftDoubleReal* kernel_output)
{
	unsigned int thread_index = blockIdx.x * blockDim.x + threadIdx.x;
	const int size = image_width * image_height;
	
	if (thread_index >= size)
		return;

	const int min_radius = kernel_size / 2;
	const int max_radius = kernel_size - min_radius;

	const int x = thread_index % image_width;
	const int y = thread_index / image_width;

	const bool is_x_left = x < max_radius;
	const bool is_x_right = x >= image_width - min_radius;
	const bool is_y_up = y < max_radius;
	const bool is_y_down = y >= image_height - min_radius;

	//printf("[%d] (%d, %d) (%d %d %d %d)\n", thread_index, x, y, is_x_left, is_x_right, is_y_up, is_y_down);
	//printf("[%d] [%d %d %d %d]\n", thread_index, is_x_left && is_y_up, is_x_right && is_y_up, is_x_left && is_y_down, is_x_right && is_y_down);
	//printf("[%d] [%d]\n", thread_index, (min_radius + y) * kernel_size + (x - (image_width - min_radius)));
	if (is_x_left && is_y_up)
	{
		int offset = (min_radius + y) * kernel_size + (min_radius + x);
		//printf("[%d] %d\n", thread_index, offset);
		kernel_output[thread_index] = kernel_input[offset];
		return;
	}

	if (is_x_right && is_y_up)
	{
		int offset = (min_radius + y) * kernel_size + (x - (image_width - min_radius));
		//printf("[%d] %d\n", thread_index, offset);
		kernel_output[thread_index] = kernel_input[offset];
		return;
	}

	if (is_x_left && is_y_down)
	{
		int offset = (y - (image_height - min_radius)) * kernel_size + (min_radius + x);
		//printf("[%d] %d\n", thread_index, offset);
		kernel_output[thread_index] = kernel_input[offset];
		return;
	}

	if (is_x_right && is_y_down)
	{
		int offset = (y - (image_height - min_radius)) * kernel_size + (x - (image_width - min_radius));
		//printf("[%d] %d\n", thread_index, offset);
		kernel_output[thread_index] = kernel_input[offset];
		return;
	}

	//printf("[%d] 0\n", thread_index);
	kernel_output[thread_index] = 0.f;
}

__global__ void scale(hipfftDoubleComplex* complex, const int image_width, const int image_height, double weight)
{
	unsigned thread_index = blockIdx.x * blockDim.x + threadIdx.x;
	const int size = image_width * image_height;

	if (thread_index >= size)
		return;

	complex[thread_index].x *= weight;
	complex[thread_index].y *= weight;
}

__global__ void convert_real_to_complex(hipfftDoubleReal* real, const int image_width, const int image_height, hipfftDoubleComplex* complex)
{
	unsigned thread_index = blockIdx.x * blockDim.x + threadIdx.x;
	const int size = image_width * image_height;

	if (thread_index >= size)
		return;

	complex[thread_index].x = real[thread_index];
	complex[thread_index].y = 0.f;
}

__global__ void convert_complex_to_real(hipfftDoubleComplex* complex, const int image_width, const int image_height, hipfftDoubleReal* real)
{
	unsigned thread_index = blockIdx.x * blockDim.x + threadIdx.x;
	const int size = image_width * image_height;

	if (thread_index >= size)
		return;

	real[thread_index] = complex[thread_index].x;
}

__global__ void pointwise_product2(hipfftDoubleComplex* a, hipfftDoubleComplex* b, int size, double weight)
{
	unsigned thread_index = blockIdx.x * blockDim.x + threadIdx.x;
	if (thread_index >= size)
		return;

	a[thread_index].x = a[thread_index].x * b[thread_index].x;
	a[thread_index].y = a[thread_index].y * b[thread_index].y;
	a[thread_index].x *= weight;
	a[thread_index].y *= weight;
}

__global__ void pointwise_product(hipfftDoubleComplex* a, hipfftDoubleComplex* b, int size, double weight)
{
	unsigned thread_index = blockIdx.x * blockDim.x + threadIdx.x;
	if (thread_index >= size)
		return;

	//printf("[%d] (%f, %f) * (%f, %f) = \n", thread_index, a[thread_index].x, a[thread_index].y, b[thread_index].x, b[thread_index].y);

	double a_real_original = a[thread_index].x;
	a[thread_index].x = a[thread_index].x * b[thread_index].x - a[thread_index].y * b[thread_index].y;
	a[thread_index].y = a_real_original * b[thread_index].y + a[thread_index].y * b[thread_index].x;
	//printf("[%d] (%f, %f) = \n", thread_index, a[thread_index].x, a[thread_index].y);
	a[thread_index].x *= weight;
	a[thread_index].y *= weight;

	//printf("[%d] = (%f, %f)\n", thread_index, a[thread_index].x, a[thread_index].y);
}

__global__ void print_2d_real(hipfftDoubleReal* d_arr, int width, int height)
{
	for (int i = 0; i < height; i++)
	{
		//printf("%d: ", i);
		for (int j = 0; j < width; j++)
		{
			//printf("%4d", j);
			printf("%3.1f ", d_arr[i * width + j]);
		}
		printf("\n");
	}
	printf("\n");
}

void ConvolutionCalculator::convolution(
	const uint8_t* image_in,
	const double* kernel,
	const int image_width,
	const int image_height,
	const int kernel_width,
	const int kernel_height,
	const int bpp,
	uint8_t* image_out
)
{
	const int image_size_with_bpp = image_width * image_height * bpp;
	const int image_size = image_width * image_height;
	const int kernel_size = kernel_width * kernel_height;

	const int complex_half_size = image_width * (image_height / 2 + 1);

	uint8_t* d_int8_image = nullptr;
	checkCudaErrors(hipMalloc((void**)&d_int8_image, image_size_with_bpp * sizeof(uint8_t)));
	checkCudaErrors(hipMemcpy(d_int8_image, image_in, image_size_with_bpp * sizeof(uint8_t), hipMemcpyHostToDevice));


	hipfftDoubleReal* d_real_kernel = nullptr;
	checkCudaErrors(hipMalloc((void**)&d_real_kernel, kernel_size * sizeof(hipfftDoubleReal)));
	checkCudaErrors(hipMemcpy(d_real_kernel, kernel, kernel_size * sizeof(hipfftDoubleReal), hipMemcpyHostToDevice));

	hipfftDoubleReal* d_real_kernel_padded = nullptr;
	checkCudaErrors(hipMalloc((void**)&d_real_kernel_padded, image_size * sizeof(hipfftDoubleReal)));

	int blocksx = ceil((image_size) / 256.0f);
	dim3 threads(256);
	dim3 grid(blocksx);

	//printf("grid: %d, threads: %d\n", grid.x, threads.x);
	pad_kernel << <grid, threads >> > (d_real_kernel, image_width, image_height, kernel_width, d_real_kernel_padded);
	checkCudaErrors(hipDeviceSynchronize());

	//print_2d_real << <1, 1 >> > (d_real_kernel_padded, image_width, image_height);

	hipfftDoubleComplex* d_complex_kernel = nullptr;
	checkCudaErrors(hipMalloc((void**)&d_complex_kernel, image_size * sizeof(hipfftDoubleComplex)));
	convert_real_to_complex << <grid, threads >> > (d_real_kernel, image_width, image_height, d_complex_kernel);

	hipfftHandle plan_complex_to_complex;
	checkCudaErrors(hipfftPlan2d(&plan_complex_to_complex, image_height, image_width, HIPFFT_Z2Z));

	hipfftDoubleComplex* d_complex_kernel_frequency = nullptr;
	checkCudaErrors(hipMalloc((void**)&d_complex_kernel_frequency, image_size * sizeof(hipfftDoubleComplex)));
	hipfftExecZ2Z(plan_complex_to_complex, d_complex_kernel, d_complex_kernel_frequency, HIPFFT_FORWARD);
	checkCudaErrors(hipDeviceSynchronize());

	//scale << <grid, threads >> > (d_complex_kernel_frequency, image_width, image_height, 1.0f / (image_width * image_height));
	checkCudaErrors(hipDeviceSynchronize());

	for (int i = 0; i < bpp; i++)
	{
		hipfftDoubleReal* d_real_image = nullptr;

		checkCudaErrors(hipMalloc((void**)&d_real_image, image_size * sizeof(hipfftDoubleReal)));

		collect_data<<<grid, threads>>>(d_int8_image, bpp, i, image_size, d_real_image);

		checkCudaErrors(hipDeviceSynchronize());
		//print_2d_real << <1, 1 >> > (d_real_image, image_width, image_height);

		hipfftDoubleComplex* d_complex_image = nullptr;
		checkCudaErrors(hipMalloc((void**)&d_complex_image, image_size * sizeof(hipfftDoubleComplex)));
		convert_real_to_complex << <grid, threads >> > (d_real_image, image_width, image_height, d_complex_image);

		hipfftDoubleComplex* d_complex_image_frequency = nullptr;
		checkCudaErrors(hipMalloc((void**)&d_complex_image_frequency, image_size * sizeof(hipfftDoubleComplex)));
		checkCudaErrors(hipfftExecZ2Z(plan_complex_to_complex, d_complex_image, d_complex_image_frequency, HIPFFT_FORWARD));
		checkCudaErrors(hipDeviceSynchronize());

		scale << <grid, threads >> > (d_complex_image_frequency, image_width, image_height, 1.0f / (image_width * image_height));
		checkCudaErrors(hipDeviceSynchronize());


		//std::cout << multiplication_grid.x << multiplication_grid.y << multiplication_grid.z << multiplication_threads.x << multiplication_threads.y << multiplication_threads.z;
		//printf("%f", 1.0f / (image_width * image_height));
		pointwise_product2 << <grid, threads>> > (d_complex_image_frequency, d_complex_kernel_frequency, image_size, 1.0f);
		checkCudaErrors(hipDeviceSynchronize());

		checkCudaErrors(hipfftExecZ2Z(plan_complex_to_complex, d_complex_image_frequency, d_complex_image, HIPFFT_BACKWARD));
		checkCudaErrors(hipDeviceSynchronize());

		convert_complex_to_real << <grid, threads >> > (d_complex_image, image_width, image_height, d_real_image);
		checkCudaErrors(hipDeviceSynchronize());

		//print_2d_real << <1, 1 >> > (d_real_image, image_width, image_height);

		write_data_to_image << <grid, threads >> > (d_int8_image, bpp, i, image_size, d_real_image);
		checkCudaErrors(hipDeviceSynchronize());

		hipFree(d_real_image);
		hipFree(d_complex_image);
		hipFree(d_complex_image_frequency);
	}
	hipfftDestroy(plan_complex_to_complex);

	checkCudaErrors(hipMemcpy(image_out, d_int8_image, image_size_with_bpp * sizeof(uint8_t), hipMemcpyDeviceToHost));

	hipFree(d_real_kernel);
	hipFree(d_real_kernel_padded);
	hipFree(d_complex_kernel);
	hipFree(d_complex_kernel_frequency);
	hipFree(d_int8_image);

	checkCudaErrors(hipPeekAtLastError());
}
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <cmath>
#include <algorithm>
#include <iostream>
#include "hipfft/hipfft.h"
#include "hipfft/hipfftXt.h"

#include "ConvolutionOld.cuh"

__global__ void write_data_to_image(uint8_t* image, int bpp, int offset, int size, hipfftReal* data);
__global__ void collect_data(uint8_t* image_in, int bpp, int offset, int size, hipfftReal* image_out);
__global__ void pad_kernel(
	hipfftReal* kernel_input,
	const int image_width,
	const int image_height,
	const int kernel_size,
	hipfftReal* kernel_output);
__global__ void pointwise_product(hipfftComplex* a, hipfftComplex* b, int size, float weight);

static const char* _cudaGetErrorEnum(hipfftResult error) {
	switch (error) {
	case HIPFFT_SUCCESS:
		return "HIPFFT_SUCCESS";

	case HIPFFT_INVALID_PLAN:
		return "HIPFFT_INVALID_PLAN";

	case HIPFFT_ALLOC_FAILED:
		return "HIPFFT_ALLOC_FAILED";

	case HIPFFT_INVALID_TYPE:
		return "HIPFFT_INVALID_TYPE";

	case HIPFFT_INVALID_VALUE:
		return "HIPFFT_INVALID_VALUE";

	case HIPFFT_INTERNAL_ERROR:
		return "HIPFFT_INTERNAL_ERROR";

	case HIPFFT_EXEC_FAILED:
		return "HIPFFT_EXEC_FAILED";

	case HIPFFT_SETUP_FAILED:
		return "HIPFFT_SETUP_FAILED";

	case HIPFFT_INVALID_SIZE:
		return "HIPFFT_INVALID_SIZE";

	case HIPFFT_UNALIGNED_DATA:
		return "HIPFFT_UNALIGNED_DATA";

	case HIPFFT_INCOMPLETE_PARAMETER_LIST:
		return "HIPFFT_INCOMPLETE_PARAMETER_LIST";

	case HIPFFT_INVALID_DEVICE:
		return "HIPFFT_INVALID_DEVICE";

	case HIPFFT_PARSE_ERROR:
		return "HIPFFT_PARSE_ERROR";

	case HIPFFT_NO_WORKSPACE:
		return "HIPFFT_NO_WORKSPACE";

	case HIPFFT_NOT_IMPLEMENTED:
		return "HIPFFT_NOT_IMPLEMENTED";

	case CUFFT_LICENSE_ERROR:
		return "CUFFT_LICENSE_ERROR";

	case HIPFFT_NOT_SUPPORTED:
		return "HIPFFT_NOT_SUPPORTED";
	}

	return "<unknown>";
}

static const char* _cudaGetErrorEnum(hipError_t error) {
	return hipGetErrorName(error);
}

template <typename T>
void check(T result, char const* const func, const char* const file,
	int const line) {
	if (result) {
		fprintf(stderr, "CUDA error at %s:%d code=%d(%s) \"%s\" \n", file, line,
			static_cast<unsigned int>(result), _cudaGetErrorEnum(result), func);
		exit(EXIT_FAILURE);
	}
}
#define checkCudaErrors(val) check((val), #val, __FILE__, __LINE__)

__global__ void collect_data(uint8_t* image_in, int bpp, int offset, int size, hipfftReal* image_out)
{
	unsigned int thread_index = blockIdx.x * blockDim.x + threadIdx.x;

	if (thread_index >= size)
		return;

	image_out[thread_index] = image_in[thread_index * bpp + offset];
}

__global__ void write_data_to_image(uint8_t* image, int bpp, int offset, int size, hipfftReal* data)
{
	unsigned int thread_index = blockIdx.x * blockDim.x + threadIdx.x;

	if (thread_index > size)
		return;

	if (data[thread_index] < 0.f)
		data[thread_index] = 0.f;

	if (data[thread_index] > 255.f)
		data[thread_index] = 255.f;
	image[thread_index * bpp + offset] = data[thread_index];
}

__global__ void pad_kernel(hipfftReal* kernel_input, const int image_width, const int image_height, const int kernel_size, hipfftReal* kernel_output)
{
	unsigned int thread_index = blockIdx.x * blockDim.x + threadIdx.x;
	const int size = image_width * image_height;
	
	if (thread_index >= size)
		return;

	const int min_radius = kernel_size / 2;
	const int max_radius = kernel_size - min_radius;

	const int x = thread_index % image_width;
	const int y = thread_index / image_width;

	const bool is_x_left = x < max_radius;
	const bool is_x_right = x >= image_width - min_radius;
	const bool is_y_up = y < max_radius;
	const bool is_y_down = y >= image_height - min_radius;

	//printf("[%d] (%d, %d) (%d %d %d %d)\n", thread_index, x, y, is_x_left, is_x_right, is_y_up, is_y_down);
	//printf("[%d] [%d %d %d %d]\n", thread_index, is_x_left && is_y_up, is_x_right && is_y_up, is_x_left && is_y_down, is_x_right && is_y_down);
	//printf("[%d] [%d]\n", thread_index, (min_radius + y) * kernel_size + (x - (image_width - min_radius)));
	if (is_x_left && is_y_up)
	{
		int offset = (min_radius + y) * kernel_size + (min_radius + x);
		//printf("[%d] %d\n", thread_index, offset);
		kernel_output[thread_index] = kernel_input[offset];
		return;
	}

	if (is_x_right && is_y_up)
	{
		int offset = (min_radius + y) * kernel_size + (x - (image_width - min_radius));
		//printf("[%d] %d\n", thread_index, offset);
		kernel_output[thread_index] = kernel_input[offset];
		return;
	}

	if (is_x_left && is_y_down)
	{
		int offset = (y - (image_height - min_radius)) * kernel_size + (min_radius + x);
		//printf("[%d] %d\n", thread_index, offset);
		kernel_output[thread_index] = kernel_input[offset];
		return;
	}

	if (is_x_right && is_y_down)
	{
		int offset = (y - (image_height - min_radius)) * kernel_size + (x - (image_width - min_radius));
		//printf("[%d] %d\n", thread_index, offset);
		kernel_output[thread_index] = kernel_input[offset];
		return;
	}

	//printf("[%d] 0\n", thread_index);
	kernel_output[thread_index] = 0.f;
}

__global__ void pointwise_product(hipfftComplex* a, hipfftComplex* b, int size, float weight)
{
	unsigned thread_index = blockIdx.x * blockDim.x + threadIdx.x;
	if (thread_index >= size)
		return;

	//printf("[%d] (%f, %f) * (%f, %f) = \n", thread_index, a[thread_index].x, a[thread_index].y, b[thread_index].x, b[thread_index].y);

	float a_real_original = a[thread_index].x;
	a[thread_index].x = a[thread_index].x * b[thread_index].x - a[thread_index].y * b[thread_index].y;
	a[thread_index].y = a_real_original * b[thread_index].y + a[thread_index].y * b[thread_index].x;
	//printf("[%d] (%f, %f) = \n", thread_index, a[thread_index].x, a[thread_index].y);
	a[thread_index].x *= weight;
	a[thread_index].y *= weight;

	//printf("[%d] = (%f, %f)\n", thread_index, a[thread_index].x, a[thread_index].y);
}

__global__ void correct_consistency(hipfftComplex* complex, const int real_width, const int real_height)
{
	//printf("\n correcting consistency: original value was: \n");
	//printf("complex : %f\n", complex[0].y);
	complex[0].y = 0.f;
	const int size = real_width * real_height;
	if (size % 2 == 0)
	{
	//	printf("complex : %f\n", complex[size / 2].y);
		complex[size / 2].y = 0.f;


	}
}

__global__ void print_2d_complex(hipfftComplex* d_arr, int width, int height)
{
	for (int i = 0; i < height; i++)
	{
		//printf("%d: ", i);
		for (int j = 0; j < width; j++)
		{
			//printf("%4d", j);
			printf("(%3.1f, %3.1f) ", d_arr[i * width + j].x, d_arr[i * width + j].y);
		}
		printf("\n");
	}
	printf("\n");
}

__global__ void print_2d_real(hipfftReal* d_arr, int width, int height)
{
	for (int i = 0; i < height; i++)
	{
		//printf("%d: ", i);
		for (int j = 0; j < width; j++)
		{
			//printf("%4d", j);
			printf("%2.3f ", d_arr[i * width + j]);
		}
		printf("\n");
	}
	printf("\n");
}

void ConvolutionCalculator::convolution_cufft(
	const uint8_t* image_in,
	const float* kernel,
	const int image_width,
	const int image_height,
	const int kernel_width,
	const int kernel_height,
	const int bpp,
	uint8_t* image_out
)
{
	const int image_size = image_width * image_height * bpp;
	const int image_real_size = image_width * image_height;
	const int kernel_size = kernel_width * kernel_height;

	const int complex_size = (image_width / 2 + 1) * image_height;

	uint8_t* d_int8_image = nullptr;
	checkCudaErrors(hipMalloc((void**)&d_int8_image, image_size * sizeof(uint8_t)));
	//create_in_device<uint8_t> << <1, 1 >> > (&d_int8_image, image_size);
	checkCudaErrors(hipMemcpy(d_int8_image, image_in, image_size * sizeof(uint8_t), hipMemcpyHostToDevice));


	hipfftReal* d_real_kernel = nullptr;
	checkCudaErrors(hipMalloc((void**)&d_real_kernel, kernel_size * sizeof(hipfftReal)));

	checkCudaErrors(hipMemcpy(d_real_kernel, kernel, kernel_size * sizeof(hipfftReal), hipMemcpyHostToDevice));

	hipfftReal* d_real_kernel_padded = nullptr;
	checkCudaErrors(hipMalloc((void**)&d_real_kernel_padded, image_real_size * sizeof(hipfftReal)));

	checkCudaErrors(hipDeviceSynchronize());

	int blocksx = ceil((image_real_size) / 256.0f);
	dim3 threads(256);
	dim3 grid(blocksx);

	//printf("grid: %d, threads: %d\n", grid.x, threads.x);
	pad_kernel << <grid, threads >> > (d_real_kernel, image_width, image_height, kernel_width, d_real_kernel_padded);
	
	checkCudaErrors(hipDeviceSynchronize());

	//print_2d_real << <1, 1 >> > (d_real_kernel_padded, image_width, image_height);

	checkCudaErrors(hipDeviceSynchronize());
	hipfftHandle plan_kernel_to_complex;
	checkCudaErrors(hipfftPlan2d(&plan_kernel_to_complex, image_height, image_width, HIPFFT_R2C));

	hipfftComplex* d_complex_kernel = nullptr;
	checkCudaErrors(hipMalloc((void**)&d_complex_kernel, complex_size * sizeof(hipfftComplex)));
	hipfftExecR2C(plan_kernel_to_complex, d_real_kernel_padded, d_complex_kernel);
	checkCudaErrors(hipDeviceSynchronize());

	for (int i = 0; i < bpp; i++)
	{
		hipfftReal* d_real_image = nullptr;

		checkCudaErrors(hipMalloc((void**)&d_real_image, image_real_size * sizeof(hipfftReal)));

		collect_data << <grid, threads >> > (d_int8_image, bpp, i, image_real_size, d_real_image);
		checkCudaErrors(hipDeviceSynchronize());

		hipfftComplex* d_complex_image = nullptr;

		checkCudaErrors(hipMalloc((void**)&d_complex_image, complex_size * sizeof(hipfftComplex)));
		checkCudaErrors(hipMemset(d_complex_image, 0, complex_size * sizeof(hipfftComplex)));
		checkCudaErrors(hipDeviceSynchronize());

		hipfftHandle plan_image_to_complex, plan_result_to_real;
		checkCudaErrors(hipfftPlan2d(&plan_image_to_complex, image_height, image_width, HIPFFT_R2C));
		checkCudaErrors(hipfftPlan2d(&plan_result_to_real, image_height, image_width, HIPFFT_C2R));

		checkCudaErrors(hipfftExecR2C(plan_image_to_complex, d_real_image, d_complex_image));
		checkCudaErrors(hipDeviceSynchronize());

		int multiplication_blocksx = ceil(complex_size / 256.0f);
		dim3 multiplication_threads(256);
		dim3 multiplication_grid(multiplication_blocksx);
		//std::cout << multiplication_grid.x << multiplication_grid.y << multiplication_grid.z << multiplication_threads.x << multiplication_threads.y << multiplication_threads.z;
		//printf("%f", 1.0f / (image_width * image_height));
		pointwise_product << <multiplication_grid, multiplication_threads >> > (d_complex_image, d_complex_kernel, complex_size, 1.0f / (image_width * image_height));
		checkCudaErrors(hipDeviceSynchronize());

		//print_2d_complex << <1, 1 >> > (d_complex_image, image_width, image_height);

		//correct_consistency << <1, 1 >> > (d_complex_image, image_width, image_height);

		checkCudaErrors(hipfftExecC2R(plan_result_to_real, d_complex_image, d_real_image));
		checkCudaErrors(hipDeviceSynchronize());

		//print_2d_real << <1, 1 >> > (d_real_image, image_width, image_height);

		write_data_to_image << <grid, threads >> > (d_int8_image, bpp, i, image_real_size, d_real_image);
		checkCudaErrors(hipDeviceSynchronize());

		hipFree(d_real_image);
		hipFree(d_complex_image);

		hipfftDestroy(plan_image_to_complex);
		hipfftDestroy(plan_result_to_real);
	}
	hipfftDestroy(plan_kernel_to_complex);

	checkCudaErrors(hipMemcpy(image_out, d_int8_image, image_size * sizeof(uint8_t), hipMemcpyDeviceToHost));

	hipFree(d_real_kernel);
	hipFree(d_real_kernel_padded);
	hipFree(d_complex_kernel);
	hipFree(d_int8_image);
	//destroy_in_device << <1, 1 >> > (d_real_kernel);
	//destroy_in_device << <1, 1 >> > (d_real_kernel_padded);
	//destroy_in_device << <1, 1 >> > (d_complex_kernel);
	//destroy_in_device << <1, 1 >> > (d_int8_image);

	checkCudaErrors(hipPeekAtLastError());
}

__device__ int d_max(int a, int b)
{
	if (a > b)
		return a;
	return b;
}

__device__ int d_min(int a, int b)
{
	if (a > b)
		return b;
	return a;
}


__global__ void conv_naive(
	const uint8_t* input, 
	const float* kernel, 
	const int width, 
	const int height,
	const int kernel_size, 
	const int bpp, 
	const int channel, 
	uint8_t* output) {
	float sum = 0;
	//printf("%d\n", width * height * bpp);
	const int size = width * height;

	unsigned thread_index = blockIdx.x * blockDim.x + threadIdx.x;

	const int x = thread_index % width;
	const int y = thread_index / width;

	if (thread_index >= size)
		return;

	//printf("[%d]((%d, %d))\n", thread_index, x, y);

	for (int k_row = - kernel_size / 2; k_row <= kernel_size / 2; k_row++) {
		for (int k_col = -kernel_size / 2; k_col <= kernel_size / 2; k_col++) {
			int offset = (k_row + kernel_size / 2) * kernel_size + k_col + kernel_size / 2;
			int x_index = x + k_col;
			if (x_index < 0 || x_index >= width)
				continue;

			int y_index = y + k_row;
			if (y_index < 0 || y_index >= height)
				continue;

			int image_offset = x_index + y_index * width;
			//if (image_offset < 0 || image_offset >= size)
			//	continue;
			
			sum += kernel[offset] * (float)(input[image_offset * bpp + channel]);
		}
	}
	output[thread_index * bpp + channel] = sum;
}

void ConvolutionCalculator::convolution_naive(
	const uint8_t* image_in, 
	const float* kernel, 
	const int image_width, 
	const int image_height, 
	const int kernel_width, 
	const int kernel_height, 
	const int bpp, 
	uint8_t* image_out)
{
	uint8_t* d_input_image = nullptr;
	uint8_t* d_output_image = nullptr;
	float* d_kernel = nullptr;
	const int image_size = image_width * image_height;

	hipMalloc((void**)&d_input_image, sizeof(uint8_t) * image_size * bpp);
	hipMalloc((void**)&d_output_image, sizeof(uint8_t) * image_size * bpp);
	hipMalloc((void**)&d_kernel, sizeof(float) * kernel_width * kernel_height);

	//printf("ImageSize: %d", image_size * bpp);

	checkCudaErrors(hipMemcpy(d_input_image, image_in, sizeof(uint8_t) * image_size * bpp, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_kernel, kernel, sizeof(float) * kernel_width * kernel_height, hipMemcpyHostToDevice));

	int blocksx = ceil((image_size) / 256.0f);
	dim3 threads(256);
	dim3 grid(blocksx);

	for (int i = 0; i < bpp; i++) {
		conv_naive << <grid, threads>> > (d_input_image, d_kernel, image_width, image_height, kernel_width, bpp, i, d_output_image);
	}

	checkCudaErrors(hipDeviceSynchronize());
	hipMemcpy(image_out, d_output_image, sizeof(uint8_t) * image_height * image_width * bpp, hipMemcpyDeviceToHost);

	hipFree(d_input_image);
}